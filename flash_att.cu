#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cmath>
#include <hiprand.h>
#include <iostream>
#include <cmath>

__global__ void forward_kernel(const float* Q, const float* K, const float* V, float* O, float* l, float* m,
                    const int B, const int H, const int N, const int d, 
                    const int Tc, const int Tr, const int Bc, const int Br, const float innerprod_scale){

    int b = blockIdx.x;    // handles b index
    int h = blockIdx.y;    // handles head index
    int br = threadIdx.x; // thread handles br dimension
    int bc = threadIdx.y;   // thread handles bc dimension
    int bz = gridDim.x;    // batch_size
    int nh = gridDim.y;    // num_head
    
    // allocate SRAM partitions
    extern __shared__ float sram[];
    float* Qi = sram;
    float* Kj = &sram[Br*d];
    float* Vj = &sram[Br*d + Bc*d];
    float* Oi = &sram[Br*d + 2*Bc*d];
    float* li = &sram[2*Br*d + 2*Bc*d];
    float* mi = &sram[2*Br*d + 2*Bc*d + Br];
    float* Sij = &sram[2*Br*d + 2*Bc*d + 2*Br];


    // modify this code to include loops of d instead of bc;

    for(int j = 0; j < Tc; j++) {
        // Load Kj, Vj blocks from HBM to SRAM
        Kj[brc*d + k] = K[b*nh*N*d + h*N*d + j*Bc*d + brc*d + k];
        Vj[brc*d + k] = V[b*nh*N*d + h*N*d + j*Bc*d + brc*d + k];
        __syncthreads();
        
        for(int i = 0; i < Tr; i++) {
            // Load Qi, Oi, li, mi from HBM to SRAM
            Qi[brc*d + k] = Q[b*nh*N*d + h*N*d + i*Br*d + brc*d + k];
            Oi[brc*d + k] = O[b*nh*N*d + h*N*d + i*Br*d + brc*d + k];
            li[brc] = l[i*Br + brc];
            mi[brc] = m[i*Br + brc];
            __syncthreads();
            
            // Sij = QiKjT, mij = rowmax(Sij), mi_new = max(mi,mij)
            float mij = -INFINITY;
            float inner_prod = 0.0f;
            for(int jj = 0; jj < Bc; jj++) {
                inner_prod += Qi[brc*d + kk] * Kj[jj*d + kk];
            }
            Sij[br*Bc+bc] = inner_prod;
            mij = fmaxf(mij, Sij[br*Bc+bc]);


            // Pij = exp(Sij - mij), lij = rowsum(Pij)
            float lij = 0.0f;
            for(int jj = 0; jj < Bc; jj++) {
                float exp_val = __expf(Sij[brc*Bc + jj] - mij);
                Sij[brc*Bc + jj] = exp_val;
                lij += exp_val;
            }
            
            // compute mi_new, li_new
            float mi_new = fmaxf(mi[brc], mij);
            float li_new = __expf(mi[brc] - mi_new) * li[brc] + __expf(mij - mi_new) * lij;
            
            // Write O, l, m to HBM
            float PijVj = 0.0f;
            for(int jj = 0; jj < Bc; jj++) {
                float inner_prod = 0.0f;
                for(int kk = 0; kk < d; kk++) {
                    inner_prod += Sij[brc*Bc + jj] * Vj[jj*d + kk];
                }
                PijVj += inner_prod;
            }
            
            O[b*nh*N*d + h*N*d + i*Br*d + brc*d + k] = 
                (1.0f / li_new) * (li[brc] * __expf(mi[brc] - mi_new) * Oi[brc*d + k] + 
                                  __expf(mij - mi_new) * PijVj);
            
            if(k == 0) {
                l[i*Br + brc] = li_new;
                m[i*Br + brc] = mi_new;
            }
            __syncthreads();
        }
        __syncthreads();
    }
}

torch::Tensor forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V, torch::Tensor O, torch::Tensor l, torch::Tensor m, 
        const int bz, const int nh, const int N, const int d) {

    const int Br = 32;
    const int Bc = 32;

    const int Tr = (N + Br - 1) / Br;
    const int Tc = (N + Bc - 1) / Bc;

    const float dot_prod_scale = 1/sqrt(d);

    dim3 grid_size(bz, nh);
    dim3 block_size(Br, d);

    // Calculate shared memory size
    const int shared_mem_size = (2*Br*d + 2*Bc*d + 2*Br + Br*Bc) * sizeof(float);

    // Check shared memory size
    //int max_sram_size;
    //hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    //printf("Max shared memory: %d, requested shared memory: %d\n", max_sram_size, shared_mem_size);

    forward_kernel<<<grid_size, block_size, shared_mem_size>>>(
        Q.data_ptr<float>(),
        K.data_ptr<float>(),
        V.data_ptr<float>(),
        O.data_ptr<float>(),
        l.data_ptr<float>(),
        m.data_ptr<float>(),
        bz, nh, N, d, Tc, Tr, Bc, Br, dot_prod_scale
    );

    return O;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Flash Attention forward");
}
